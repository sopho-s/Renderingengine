#include "AMaths.cuh"
#include "GMaths.cuh"
#include "Tester.h"
#include <iostream>
#include <stdlib.h>

void EulerToQuaternion() {
	AMaths::Euler testval;
	testval.yaw = 1;
	testval.pitch = 2;
	testval.roll = 1;
	AMaths::Quaternion quat = AMaths::EulerToQuaternion(testval);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.w, 0.610, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.i, -0.127, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.j, 0.772, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.k, -0.127, 0.001);
}
void QuaternionToEuler() {
	AMaths::Quaternion testval;
	testval.w = 0.610;
	testval.i = -0.127;
	testval.j = 0.772;
	testval.k = -0.127;
	AMaths::Euler eul = AMaths::QuaternionToEuler(testval);
	Tester::ASSERT_NEAR_EQUAL<float>(eul.yaw, -2.1392, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(eul.pitch, 1.1423, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(eul.roll, -2.1392, 0.001);
}
void EQ() {
	AMaths::Euler eul;
	eul.yaw = 9;
	eul.pitch = 22;
	eul.roll = 1;
	AMaths::Quaternion quat = AMaths::EulerToQuaternion(eul);
}
void QE() {
	AMaths::Quaternion quat;
	quat.w = 0.949;
	quat.i = 0.133;
	quat.j = 0.169;
	quat.k = 0.231;
	AMaths::Euler eul = AMaths::QuaternionToEuler(quat);
}

void QEQ() {
	AMaths::Quaternion testval;
	testval.w = 0.610;
	testval.i = -0.127;
	testval.j = 0.772;
	testval.k = -0.127;
	AMaths::Quaternion truth;
	truth.w = 0.610;
	truth.i = -0.127;
	truth.j = 0.772;
	truth.k = -0.127;
	AMaths::Euler eul = AMaths::QuaternionToEuler(testval);
	AMaths::Quaternion quat = AMaths::EulerToQuaternion(eul);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.w, truth.w, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.i, truth.i, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.j, truth.j, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(quat.k, truth.k, 0.001);
}

void QuaternionMultiplication() {
	AMaths::Quaternion inp1;
	inp1.w = 2;
	inp1.i = 1;
	inp1.j = 3;
	inp1.k = -1;
	AMaths::Quaternion inp2;
	inp2.w = 2;
	inp2.i = -0.1;
	inp2.j = -2;
	inp2.k = 9;
	AMaths::Quaternion out;
	AMaths::QuaternionMultiplication(inp1, inp2, out);
	Tester::ASSERT_NEAR_EQUAL<float>(out.w, 19.1, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(out.i, 26.8, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(out.j, -6.9, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(out.k, 14.3, 0.001);
}

void Rotation() {
	AMaths::Quaternion testval;
	testval.w = 0.707;
	testval.i = 0;
	testval.j = 0.707;
	testval.k = 0;
	AMaths::Vector3 out;
	out.x = 1;
	out.y = 0;
	out.z = 0;
	AMaths::Quaternion vecquat;
	AMaths::Vector3ToQuaternion(out, vecquat);
	AMaths::Quaternion inverseout;
	AMaths::InverseQuaternion(testval, inverseout);
	AMaths::Quaternion result1;
	AMaths::QuaternionMultiplication(testval, vecquat, result1);
	AMaths::Quaternion result2;
	AMaths::QuaternionMultiplication(result1, inverseout, result2);
	AMaths::QuaternionToVector3(result2, out);
	Tester::ASSERT_NEAR_EQUAL<float>(out.x, 0, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(out.y, 0, 0.001);
	Tester::ASSERT_NEAR_EQUAL<float>(out.z, -1, 0.001);
}
void VecAddCPU() {
	srand (time(NULL));
	float vector1[131072];
	float vector2[131072];
	for (int i = 0; i < 131072; i++) {
		vector1[i] = rand() / RAND_MAX;
		vector2[i] = rand() / RAND_MAX;
	}
	float out[131072];
	AMaths::VecAdd(vector1, vector2, 131072, out);
}

void VecAddGPU() {
	int N = 1<<17;
	srand (time(NULL));
	float *gpuvector1, *gpuvector2, *gpuout;
	hipMallocManaged(&gpuvector1, sizeof(float) * 131072);
	hipMallocManaged(&gpuvector2, sizeof(float) * 131072);
	for (int i = 0; i < N; i++) {
		gpuvector1[i] = rand() / RAND_MAX;
		gpuvector2[i] = rand() / RAND_MAX;
	}
	hipMallocManaged(&gpuout, sizeof(float) * 131072);
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	GMaths::VecAdd <<< numBlocks, blockSize >>> (gpuvector1, gpuvector2, gpuout);
	hipFree(gpuvector1);
	hipFree(gpuvector2);
	hipFree(gpuout);
}

int main() {
	std::function<void()> test1 = [] { 
		EulerToQuaternion();
	};
	std::function<void()> test2 = [] { 
		QuaternionToEuler();
	};
	std::function<void()> test3 = [] { 
		QEQ();
	};
	std::function<void()> test4 = [] { 
		EQ();
	};
	std::function<void()> test5 = [] { 
		QE();
	};
	std::function<void()> test6 = [] { 
		QuaternionMultiplication();
	};
	std::function<void()> test7 = [] { 
		Rotation();
	};
	std::function<void()> test8 = [] { 
		VecAddGPU();
	};
	std::function<void()> test9 = [] {
		VecAddCPU();
	};
	Tester::Tester tester = Tester::Tester();
	tester.AddGroup("Angles");
	tester.AddTest(test1, "Euler To Quaternion");
	tester.AddTest(test2, "Quaternion To Euler");
	tester.AddTest(test3, "Quaternion To Euler To Quaternion");
	tester.AddAverageTimeTest(test4, 1000, "Euler To Quaternion");
	tester.AddAverageTimeTest(test5, 1000, "Quaternion To Euler");
	tester.AddTest(test6, "Quaternion Multiplication");
	tester.AddTest(test7, "Vector3 Rotation");
	tester.AddAverageTimeTest(test7, 1000, "Vector3 Rotation");
	tester.AddGroup("GPU Functionality");
	tester.AddTest(test8, "Vector Add");
	tester.AddTest(test9, "Vector Add CPU");
	tester.RunTests();
	Tester::PerformanceTester testerper = Tester::PerformanceTester();
	testerper.AddGroup("GPU");
	testerper.AddAverageTest(test9, 1, "Vector Add CPU");
	testerper.AddAverageTest(test8, 1, "Vector Add GPU");
	testerper.RunTests();
}